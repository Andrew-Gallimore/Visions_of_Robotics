#include "hip/hip_runtime.h"
#include <string>
#include <vector>
#include <cmath>
#include <iostream>
#include "utils/imageUtils.h"
#include "utils/timer.h"
#include "Structs.h"

using namespace std;

__global__
void getDisparities(unsigned char* leftImage, unsigned char* rightImage, int cols, int rows, int windowSize, unsigned char* disparities, unsigned char* confidence) {
    int smallest = INT_MAX;
    int long average = 0;
    int d = 0;

    int leftX = blockDim.x * blockIdx.x + threadIdx.x;
    int leftY = blockDim.y * blockIdx.y + threadIdx.y;

    int minValue = 255;
    int maxValue = 0;
    // Sample the initial window to see if it has enough detail
    for(int x = -1 * windowSize; x < windowSize; x++) {
        for(int y = -1 * windowSize; y < windowSize; y++) {
            int index = (leftX + x * 2) + ((leftY + y * 2) * cols);
            if(index < 0 || index >= cols * rows) {
                continue;
            }
            if (leftImage[index] < minValue) {
                minValue = leftImage[index];
            }
            if (leftImage[index] > maxValue) {
                maxValue = leftImage[index];
            }
        }
    }

    if(maxValue - minValue < 30) {
        int index = (leftY * cols) + leftX;
        confidence[index] = 210;
        // confidence[index - 1] = 210;
        // confidence[index + cols] = 210;
        // confidence[index + cols - 1] = 210;
        disparities[index] = 0;
        // disparities[index - 1] = 0;
        // disparities[index + cols] = 0;
        // disparities[index + cols - 1] = 0;
        return;
    }
    

    for(int offset = 0; offset < leftX && offset < 200; offset += windowSize) {
    
        int comparison = 0;

        for(int x = -1 * windowSize; x < windowSize; x++) {
            for(int y = -1 * windowSize; y < windowSize; y++) {
                int adjX = x * 3;
                int adjY = y * 3;
                if(adjX < 2 && adjY > -2 && adjY < 2 && adjY > 2) {
                    adjX = x;
                    adjX = y;
                }
                int leftIndex = (leftX + adjX) + ((leftY + adjY) * cols);
                int rightIndex = ((leftX - offset) + adjX) + ((leftY + adjY) * cols);
                
                if(leftIndex < 0 || leftIndex >= cols * rows || rightIndex < 0 || rightIndex >=  cols * rows) {
                    continue;
                }
                
                int diff = leftImage[leftIndex] - rightImage[rightIndex];
                comparison += diff * diff;
            }
        }
        
        if(comparison < smallest) {
            smallest = comparison;
            average += smallest;
            d = offset;
        }
    }

    int index = (leftY * cols) + leftX;

    int conf = 0;

    if(maxValue - minValue < 30) {
        conf = 210;
    }

    disparities[index] = d;
    // disparities[index - 1] = d;
    // disparities[index + cols] = d;
    // disparities[index + cols - 1] = d;
    confidence[index] = conf;
    // confidence[index - 1] = conf;
    // confidence[index + cols] = conf;
    // confidence[index + cols - 1] = conf;
}

// ==== Starting Function for Algorithm ====

// Runs the whole matching algorithm
int main() {
    convertJPGToPPM("images/newLeft.jpg", "images/colorTEMP.ppm");
    convertJPGToPPM("images/newRightRect.jpg", "images/colorTEMP2.ppm");
    convertPPMToBW("images/colorTEMP.ppm", "images/bwTEMP.ppm");
    convertPPMToBW("images/colorTEMP2.ppm", "images/bwTEMP2.ppm");
    PPMImage* leftImage = readPPM("images/bwTEMP.ppm", 0);
    PPMImage* rightImage = readPPM("images/bwTEMP2.ppm", 0);

    // convertJPGToPPM("opencvrect/nvcamtest_11219_s01_00000.jpg", "images/colorTEMPL_26_april.ppm");
    // convertJPGToPPM("opencvrect/nvcamtest_11114_s00_00000.jpg", "images/colorTEMPR_26_april.ppm");

    // convertJPGToPPM("opencvrect/nvcamtest_11219_s01_00000.jpg", "images/colorTEMPL_26_april.ppm");
    // convertJPGToPPM("opencvrect/nvcamtest_11219_s01_00000.jpg", "images/colorTEMPR_26_april.ppm");

    // PPMImage* leftImage = readPPM("images/colorTEMPL_26_april.ppm", 0);
    // PPMImage* rightImage = readPPM("images/colorTEMPR_26_april.ppm", 0);

    Timer totalTimer;
    Timer kernalTimer;

    
    int windowSize;
    cout << "(4=9x9,5=11x11,etc..)\nEnter windowSize: ";
    cin >> windowSize;
    
    unsigned char disparities[leftImage->width * leftImage->height] = {0};
    unsigned char confidence[leftImage->width * leftImage->height] = {0};
    
    unsigned char* d_left; 
    unsigned char* d_right;
    unsigned char* d_disparities;
    unsigned char* d_confidence;
    
    hipMalloc((void**) &d_left, leftImage->width * leftImage->height * sizeof(unsigned char)); 
    hipMalloc((void**) &d_right, rightImage->width * rightImage->height * sizeof(unsigned char)); 
    hipMalloc((void**) &d_disparities, rightImage->width * rightImage->height * sizeof(unsigned char)); 
    hipMalloc((void**) &d_confidence, rightImage->width * rightImage->height * sizeof(unsigned char)); 

    totalTimer.start();

    hipMemcpy(d_left, leftImage->data, leftImage->width * leftImage->height * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(d_right, rightImage->data, rightImage->width * rightImage->height * sizeof(unsigned char), hipMemcpyHostToDevice);
    
    int BLOCK_SIZE = 16;
    
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((leftImage->width + BLOCK_SIZE - 1) / BLOCK_SIZE, (leftImage->height + BLOCK_SIZE - 1) / BLOCK_SIZE);
    
    kernalTimer.start();
    
    getDisparities<<<grid, block>>>(d_left, d_right, leftImage->width, leftImage->height, windowSize, d_disparities, d_confidence);
    hipDeviceSynchronize();
    
    kernalTimer.stop();

    hipMemcpy(disparities, d_disparities, leftImage->width * leftImage->height * sizeof(unsigned char), hipMemcpyDeviceToHost);
    hipMemcpy(confidence, d_confidence, leftImage->width * leftImage->height * sizeof(unsigned char), hipMemcpyDeviceToHost);
    
    
    writePPM("images/depthMapP.ppm", leftImage->width, leftImage->height, 255, 0, disparities);
    writePPM("images/confidenceP.ppm", leftImage->width, leftImage->height, 255, 0, confidence);

    // Making mixed color image for the depthMap/confidenceMap.
    // Colors are interlaced in the array: r, g, b, r, g, b, etc.
    unsigned char* mixedImage = new unsigned char[leftImage->width * leftImage->height * 3];
    for (int i = 0; i < leftImage->width * leftImage->height; i++) {
        if (confidence[i] > 0) {
            // No confidence: black
            mixedImage[i * 3 + 0] = 0;   // Red
            mixedImage[i * 3 + 1] = 0;   // Green
            mixedImage[i * 3 + 2] = 0;   // Blue
        } else {
            // Confidence: red-blue gradient based on disparity
            if (disparities[i] < 10) { // Below the range, continue blue
                mixedImage[i * 3 + 0] = 0;             // Red
                mixedImage[i * 3 + 1] = 0;             // Green
                mixedImage[i * 3 + 2] = 50;           // Blue
            } else if (disparities[i] > 90) { // Above the range, continue red
                mixedImage[i * 3 + 0] = 255;           // Red
                mixedImage[i * 3 + 1] = 0;             // Green
                mixedImage[i * 3 + 2] = 0;             // Blue
            } else { // Within the range, scale red to blue
                float normalized = (disparities[i] - 10) / 80.0f; // Normalize to range [0, 1]
                mixedImage[i * 3 + 0] = static_cast<unsigned char>(normalized * 255); // Red
                mixedImage[i * 3 + 1] = 0;                                           // Green
                mixedImage[i * 3 + 2] = static_cast<unsigned char>((1.0f - normalized) * 50); // Blue
            }
        }
    }
    writePPM("images/mix.ppm", leftImage->width, leftImage->height, 255, 1, mixedImage);
    delete[] mixedImage;
    
    totalTimer.stop();

    // Got from running calibration on the images
    // fx, 0,  Ox
    // 0,  fy, Oy
    // 0,  0,  1
    float calibMatrixLeft[9] = {
        //542.131, 0, 256.252,
        //0, 720.9599, 274.971,
        //0, 0, 1
        561.85034, 0.00000, 351.88312,
        0.00000, 763.06970, 200.38995,
        0.00000, 0.00000, 1.00000
    };
    float calibMatrixRight[9] = {
        //566.5176, 0, 252.2899,
        //0, 753.3832, 218.9708,
        //0, 0, 1
        560.63837, 0.00000, 377.36542,
        0.00000, 750.10541, 200.71365,
        0.00000, 0.00000, 1.00000
    };

    printf("\nKernal time: %d ms\n", (int)kernalTimer.elapsedMilliseconds());
    printf("Total time: %d ms\n", (int)totalTimer.elapsedMilliseconds());

    return 0;
}
